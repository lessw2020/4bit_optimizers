#include "hip/hip_runtime.h"
// Cuda kernels for quant packaging (quant and dequant)

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <torch/extension.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>
#include <THC/THCAtomics.cuh>

using torch::Tensor;

using namespace std;

Tensor pack_nonlinear_cuda(Tensor data,
                            Tensor qmap) {
    return pack_nonlinear_4bit_cuda(data, qmap)

                            }

Tensor pack_nonlinear_4bit_cuda(Tensor data,  Tensor qmap)
{
    bits = 4  // hardcoding this as we are only doing 4 bit
    int64_t num_groups = data.size(0);
    int64_t group_size = data.size(1);

    // calc total bits
    const int work_per_int = 8 /bits;
    const int workint_per_thread = 4;
    const int work_per_thread = work_per_int * workint_per_thread;

    TORCH_CHECK(8 % bits ==0);

    TORCH_CHECK(group_size % work_per_thread ==0);

    int64_t total_bits = (int64_t)bits * (num_groups * group_size);
    auto options = torch::TensorOptions().type(torch::kInt8).device(data.device());
    Tensor packed = torch::empty({(total_bits +8)/8,}, options);

    // Random numbers
    int threads = group_size;
    auto gen = at::check_generator<at::CUDAGeneratorImpl>(at::cuda::detail::getDefaultCUDAGenerator());
    pair<uint64_t, uint64_t> rng_engine_inputs;
    {
        std::lock_guard<mutex> lock(gen->mutex_);
        rng_engine_inputs = gen->philox_engine_inputs(threads * work_per_thread);

    }

    // Call packing kernel
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(data.scalar_type(), "pack_nonlinear_4bit", ([&])){
        pack_nonlinear_4bit_kernel<scalar_t, false><<<num_groups, group_size / work_per_thread>>>(
            data.data_ptr<scalar_t>(),
            qmap.data_ptr<float>(),
            packed.data_ptr<int8_t>(),
            rng_engine_inputs);
    }));
        
    }

}

// pack 16/32 bit data into int8 bit stream, bits ==4
template<typename scalar_t>
__global__ void pack_nonlinear_4bit_kernel(const scalar_t* __restrict__ data, 
const float* __restrict__ qmap, 
int8_t* __restrict__ packed,
pair<uint64_t, uint64_t> seeds) {
    const int bits = 4;

    const int group_id = blockIdx.x;
    const int id_in_group = threadIdx.x;
    const int64_t global_id = group_id * blockDim.x + id_in_group;
    const int work_per_int = 8 / bits;
    const int workint_per_thread = 4;
    const int work_per_thread = work_per_int << 2; // mul by 4...
    const int8_t mask = (1 << bits) - 1;
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seeds, first, global_id, seeds.second, &state);

    for (int i = 0; i < workint_per_thread; i++) {
        uint8_t local_packed = 0;
        int64_t packed_id = global_id * workint_per_thread + i;
        for (int j = 0; j < work_per_int; j++) {
            const int64_t data_id = global_id * work_per_thread + i * work_per_int + j;
            const float noise = hiprand_uniform(&state)
            const uint8_t qx = (uint8_t)quantize_bsearch(qmap, x, noise);
            local_packed |= ((qx & mask) << (j * bits));
        }
        packed[packed_id] = local_packed;
    }
}
